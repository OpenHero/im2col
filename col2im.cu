#include "hip/hip_runtime.h"
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include "common.h"

template <typename Dtype>
__global__ void col2im_gpu_kernel(const int n, const Dtype* data_col,
    const int height, const int width, const int channels, const int ksize,
    const int pad, const int stride, const int height_col, const int width_col,
    Dtype* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = 0;
    int w = index % width + pad;
    int h = (index / width) % height + pad;
    int c = index / (width * height);
    // compute the start and end of the output
    int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
    int w_col_end = min(w / stride + 1, width_col);
    int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
    int h_col_end = min(h / stride + 1, height_col);
    /*
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        // the col location: [c * width * height + h_out, w_out]
        int c_col = c * ksize * ksize + (h - h_col * stride) * ksize + (w - w_col * stride);
        val += data_col[(c_col * height_col + h_col) * width_col + w_col];
      }
    }
    */
    // equivalent implementation
    int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
    int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
    int coeff_w_col = (1 - stride * height_col * width_col);
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
      }
    }
    data_im[index] = val;
  }
}

//__global__ void col2im_gpu_kernel<float>(const int n, const float* data_col,
//    const int height, const int width, const int channels, const int ksize,
//    const int pad, const int stride, const int height_col, const int width_col,
//    float* data_im) {
//  CUDA_KERNEL_LOOP(index, n) {
//    float val = 0;
//    int w = index % width + pad;
//    int h = (index / width) % height + pad;
//    int c = index / (width * height);
//    // compute the start and end of the output
//    int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
//    int w_col_end = min(w / stride + 1, width_col);
//    int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
//    int h_col_end = min(h / stride + 1, height_col);
//    /*
//    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
//      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
//        // the col location: [c * width * height + h_out, w_out]
//        int c_col = c * ksize * ksize + (h - h_col * stride) * ksize + (w - w_col * stride);
//        val += data_col[(c_col * height_col + h_col) * width_col + w_col];
//      }
//    }
//    */
//    // equivalent implementation
//    int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
//    int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
//    int coeff_w_col = (1 - stride * height_col * width_col);
//    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
//      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
//        val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
//      }
//    }
//    data_im[index] = val;
//  }
//}

template <typename Dtype>
void col2im_gpu(const Dtype* data_col, const int channels,
    const int height, const int width, const int ksize, const int pad,
    const int stride, Dtype* data_im) {
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  col2im_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_col, height, width, channels, ksize, pad, stride,
      height_col, width_col, data_im);
  CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void col2im_gpu<float>(const float* data_col, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, float* data_im);
template void col2im_gpu<double>(const double* data_col, const int channels,
    const int height, const int width, const int psize, const int pad,
    const int stride, double* data_im);

// Helper function for using CUDA to add vectors in parallel.
hipError_t col2imWithCuda(float *c, float *a, float *b, unsigned int size)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;
	StopWatchInterface *timer = NULL;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	sdkCreateTimer(&timer);

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	sdkStartTimer(&timer);

    // Launch a kernel on the GPU with one thread for each element.
	dim3 block(32,16);
	dim3 grid(size/(32*16));
    //addKernel <<< grid, block>>>(dev_c, dev_a, dev_b, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

	sdkStopTimer(&timer);
	double elapsedTimeInMs = sdkGetTimerValue(&timer);
	printf("time is %fms\n", elapsedTimeInMs);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    sdkDeleteTimer(&timer);

    return cudaStatus;
}