#include "hip/hip_runtime.h"
/********************************************************************
created:	2014/07/11
created:	11:7:2014   16:22
file base:	bu_im2col
file ext:	cu
author:		Zhao Kaiyong

zhao.kaiyong(at)gmail.com
kyzhao(at)comp.hkbu.edu.hk
http://www.comp.hkbu.edu.hk/~kyzhao/
http://blog.csdn.net/openhero

purpose:
Based on caffe im2col. Merge the loop into one kernel.
On GTX640:
image 256*256 with 3 channels
batch size is 128
The time show as below:
caffe is 106.883766ms
bu_im2col is 22.095470ms	
*********************************************************************/
#include "common.h"

template <typename Dtype>
__global__ void bu_im2col_gpu_kernel(
	const int n, const Dtype* data_im,
	const int height, const int width, const int ksize, const int pad,
	const int stride, const int height_col, const int width_col,
	Dtype* data_col,
	const int data_im_size,
	const int data_col_size,
	const int batch_size) 
{
	for(int batch_index = 0; batch_index < batch_size; batch_index++)
	{
		for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < n; index += blockDim.x * gridDim.x){
			int w_out = index % width_col;
			int h_index = index / width_col;
			int h_out = h_index % height_col;
			int channel_in = h_index / height_col;
			int channel_out = channel_in * ksize * ksize;
			int h_in = h_out * stride - pad;
			int w_in = w_out * stride - pad;
			Dtype* data_col_ptr = data_col;
			data_col_ptr += batch_index* data_col_size + (channel_out * height_col + h_out) * width_col + w_out;
			const Dtype* data_im_ptr = data_im;
			data_im_ptr += batch_index* data_im_size + (channel_in * height + h_in) * width + w_in;

			Dtype temp_ret = 0.0f;
			for (int i = 0; i < ksize; ++i) {
				for (int j = 0; j < ksize; ++j) {
					int h = h_in + i;
					int w = w_in + j;
					temp_ret += (h >= 0 && w >= 0 && h < height && w < width) ?
						data_im_ptr[i * width + j]  : 0;
					data_col_ptr += height_col * width_col;
				}
			}

		}
	}
}

template <typename Dtype>
void bu_im2col_gpu(const Dtype* data_im, const int channels,
				   const int height, const int width, const int ksize, const int pad,
				   const int stride, Dtype* data_col, const int batch_size)
{
	// We are going to launch channels * height_col * width_col kernels, each
	// kernel responsible for copying a single-channel grid.
	int height_col = (height + 2 * pad - ksize) / stride + 1;
	int width_col = (width + 2 * pad - ksize) / stride + 1;
	int num_kernels = channels * height_col * width_col;

	int data_im_size = height*width*channels;
	int data_col_size = num_kernels*ksize*ksize;
	// NOLINT_NEXT_LINE(whitespace/operators)
	bu_im2col_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels), // num_kernels/16, means each thread process 16 elements
		CAFFE_CUDA_NUM_THREADS>>>(
		num_kernels, data_im, height, width, ksize, pad, stride, height_col,
		width_col, data_col, data_im_size, data_col_size, batch_size);
	CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void bu_im2col_gpu<float>(
	const float* data_im, const int channels,
	const int height, const int width, const int ksize, const int pad,
	const int stride, float* data_col,
	const int batch_size);
template void bu_im2col_gpu<double>(
	const double* data_im, const int channels,
	const int height, const int width, const int ksize, const int pad,
	const int stride, double* data_col,
	const int batch_size);


// Helper function for using CUDA to add vectors in parallel.
//const float* data_im // raw data,
//const int channels // image channels
//const int height //image height
//const int width // image width
//const int ksize // kernel size
//const int pad // pad size
//const int stride // stride size
//const int height_col // output column height
//const int width_col // output column width
//float* data_col // outpu data

hipError_t bu_im2colWithCuda(
	const float* data_im,
	const int batch_size,
	const int channels,
	const int height, const int width, const int ksize, const int pad,
	const int stride,
	float* data_col,
	const int num_kernels,
	float* data_kernel,
	float* data_ret)
{
	float *dev_a = 0;
	float *dev_c = 0;
	float *dev_k = 0;
	float *dev_ret = 0;
	hipError_t cudaStatus;
	StopWatchInterface *timer = NULL;

	hipblasHandle_t handle;
	hipblasStatus_t ret;

	ret = hipblasCreate(&handle);
	if (ret != HIPBLAS_STATUS_SUCCESS)
	{
		printf("hipblasCreate returned error code %d, line(%d)\n", ret, __LINE__);
		goto Error;
	}

	sdkCreateTimer(&timer);

	int height_col = (height + 2 * pad - ksize) / stride + 1;
	int width_col = (width + 2 * pad - ksize) / stride + 1;

	int K = ksize*ksize*channels;
	int M = num_kernels;
	int N = height_col*width_col*batch_size;

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, N * K * batch_size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, height * width * channels * batch_size* sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, data_im, height * width * channels * batch_size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_k, num_kernels*ksize*ksize*channels * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_k, data_kernel, num_kernels*ksize*ksize*channels * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_ret, num_kernels*height_col *width_col *batch_size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	const float alpha = 1.0f;
	const float beta  = 0.0f;
	int Batch_N = N * batch_size;
	sdkStartTimer(&timer);
	// Launch a kernel on the GPU with one thread for each element.
	bu_im2col_gpu<float>(dev_a, channels, height, width, ksize, pad, stride, dev_c, batch_size);
	//Perform warmup operation with cublas

	ret = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
		Batch_N , M,  K, &alpha,
		dev_c, Batch_N, dev_k, K, &beta, dev_ret, Batch_N);

	if (ret != HIPBLAS_STATUS_SUCCESS)
	{
		printf("hipblasSgemm returned error code %d, line(%d)\n", ret, __LINE__);
		goto Error;
	}
	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching im2col Kernel!\n", cudaStatus);
		goto Error;
	}

	sdkStopTimer(&timer);
	double elapsedTimeInMs = sdkGetTimerValue(&timer);
	printf("bu_im2col is %fms\n", elapsedTimeInMs);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(data_col, dev_c, N * K *batch_size* sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(data_ret, dev_ret, num_kernels*height_col *width_col *batch_size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	hipblasDestroy(handle);
Error:

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_k);
	hipFree(dev_ret);
	sdkDeleteTimer(&timer);

	return cudaStatus;
}