
#include <stdio.h>
#include <cstdlib>

#include "im2col.hpp"

void init_data(float * data, int size)
{
	for(int i = 0; i < size; i++)
	{
		data[i] = (float)rand()/(float)size;
	}
}

int main()
{
	const int height = 256;
	const int width = 256;
	const int channels = 3;
	const int batch_size = 32;//128;
	const int ksize = 5; // 5-11
	const int pad = 2; // 0-2
	const int stride = 1; // 1
	const int num_kernels = 32;

			// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

	const int arraySize = height * width * channels * batch_size; //each bacth have 128 image, each image have 256*256 size and 3 channels
	float *image;// = new float[arraySize];// = { 1, 2, 3, 4, 5 };
	checkCudaErrors(hipMallocManaged(&image,arraySize*sizeof(float)));

	int height_col = (height + 2 * pad - ksize) / stride + 1;
	int width_col = (width + 2 * pad - ksize) / stride + 1;
	int colArraySize = height_col * width_col * channels *ksize*ksize* batch_size;
	float *col1;// = new float[colArraySize]();// = { 10, 20, 30, 40, 50 };
	float *col2;// = new float[colArraySize]();// = { 0 };
	checkCudaErrors(hipMallocManaged(&col1, sizeof(float)*colArraySize));
	checkCudaErrors(hipMallocManaged(&col2, sizeof(float)*colArraySize));
	
	const int kernelArraySize = num_kernels*ksize*ksize*channels;
	float *data_kernel;// = new float[kernelArraySize];
	checkCudaErrors(hipMallocManaged(&data_kernel, sizeof(float)*kernelArraySize));

	const int resultArraySize = num_kernels * height_col * width_col*batch_size;
	float *r1;// = new float[resultArraySize]();
	float *r2;// = new float[resultArraySize]();
	checkCudaErrors(hipMallocManaged(&r1, sizeof(float)*resultArraySize));
	checkCudaErrors(hipMallocManaged(&r2, sizeof(float)*resultArraySize));

	srand(2014);
	init_data(image, arraySize);
	init_data(data_kernel, kernelArraySize);

	// image to col
	checkCudaErrors(im2colWithCuda(image, batch_size, channels, height, width, ksize, pad, stride, col1, num_kernels, data_kernel, r1));


	checkCudaErrors(bu_im2colWithCuda(image, batch_size, channels, height, width, ksize, pad, stride, col2, num_kernels, data_kernel,r2));

	int ret = -1;
	ret = check_result(col1,col2, resultArraySize);
	printf("Im2col error at %d.\n", ret);

	ret = check_result(r1,r2, resultArraySize);

	printf("Error at %d.\n", ret);

	checkCudaErrors(hipFree(image));
	checkCudaErrors(hipFree(col1));
	checkCudaErrors(hipFree(col2));
	checkCudaErrors(hipFree(data_kernel));
	checkCudaErrors(hipFree(r1));
	checkCudaErrors(hipFree(r2));

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}	
	return 0;
}

